#include "hip/hip_runtime.h"
#define USEGPU //necessary for all the inclusions to be performed correctly
//#define DOUBLE_PREC

#define NITERMAX 1000
//#include <sys/time.h>
#include <ctime>

#include <iostream>

//constants, global variables, geometric parameters et cetera 
#include "./Include/global_const.cc"
#include "./Include/global_macro.cc"
#include "./Include/global_var.cc"
#include "./Init/init.cc"
#include "./Fermions/fermions.cc"




#include "./DeviceInfo/device_info.cu"
#include "./Cuda/cuda_globalstuff.cu"
#include "./Packer/packer.cc"
#ifndef GPU_DOUBLE_PREC
 #include "./Cuda/cuda_dslash_eo.cu"
#else 
 #include "./Cuda/cuda_dslash_dd_eo.cu"
#endif

using namespace std;

int main(){

/**************************************
Matrix vector multiplication
***************************************/

  //chooseGPU(0);


//All SU(3) links set to identity
//A Random SU(3) field can be produced with init(1)
   init(1);
   cout << "Initialized Random Geuge Matrix.\n";
//If needed the gauge field can be loaded from file
//it can also be loaded using Init(2), in that case a 
//file named 'config' will be looked for.
//The gauge field can also be saved :
//gauge_conf->saveToFile("TestConf.cnf");
   Fermion* tempFermion = new Fermion();//initialized to 0
   //tempFermion->gauss(0); //produces a gaussian noise on the first component 
                            //the other components are set to constant
   cout << "Loading fermion from StartFermion.fer...\n";
   tempFermion->loadFromFile("StartFermion.fer");//choose a name

     
// GPU gauge and geometry setup
    smartpack_gauge( gauge_field_packed , gauge_conf );
    make_shift_table(shift_table);//writes the geometry info in shift_table
    chooseGPU(gpu_device_to_use);
    cuda_init_all();//moves the packed conf and shift_table onto the gpu
// END GPU gauge and geometry setup

// creating GPU arrays for the fermions
// actually for the current implementation of DslashOperatorEO()
// one would suffice

#ifndef DOUBLE_PREC
    size_t vector_size = sizeof(float2)*6*size ;//in both cases,same dimension
    float2 *Mp, *MMp;
#else
    //in this case the double precision fermion vector must be reconstructed.
    size_t vector_size = sizeof(double2)*3*size ;//in both cases,same dimension
    double2 *Mp, *MMp;
    float2 *startFermGPU;//fermion vector will initially be loaded here
#endif
   
 
    cudaSafe(AT,hipMalloc((void**)&Mp ,vector_size), "hipMalloc");
    cudaSafe(AT,hipMalloc((void**)&MMp,vector_size), "hipMalloc");
    cudaSafe(AT,hipMemset(Mp,0,vector_size),"hipMemset");
    cudaSafe(AT,hipMemset(MMp,0,vector_size),"hipMemset");
#ifdef DOUBLE_PREC    
    cudaSafe(AT,hipMalloc((void**)&startFermGPU,vector_size), "hipMalloc");
    cudaSafe(AT,hipMemset(startFermGPU,0,vector_size),"hipMemset");
#endif
// this function takes a fermion on the host 
// and puts it into an array directly on the gpu


// DslashOperatorEO called with 1 as the third argument reads the EVEN part 
// of the 2nd argument and writes the result in the ODD part of the 1st 
// DslashOperatorEO called with -1 as the third argument reads the ODD part 
// of the 2nd argument and writes the result in the EVEN part of the 1st 



#ifndef DOUBLE_PREC
   smartPackFermionOnDeviceD(MMp,tempFermion);//packing the fermion vector into
                                              // MMp
#else
   smartPackFermionOnDeviceD(startFermGPU,tempFermion);//loading fermion into
                                                       // startFermGPU
   InitR(MMp, startFermGPU ,3*size);// Initializing the double precision 
                                    // fermion vector, into MMp
#endif


//  hipEvent_t start, stop;
//  float time;
//  hipEventCreate(&start);
//  hipEventCreate(&stop);
     
//  hipEventRecord(start, 0);
    
//  struct timeval tim;  
//  gettimeofday(&tim, NULL);  
//  double t1=tim.tv_sec+(tim.tv_usec/1000000.0);  

    clock_t start, end;
    start = clock();

    for(int iter = 0; iter < NITERMAX ; iter++){
#ifndef DOUBLE_PREC
//      cout << "Multiplying by Doe, on device.\n";
        DslashOperatorEO(Mp,MMp, 1 );
//      cout << "Multiplying by Deo, on device.\n";
        DslashOperatorEO(MMp, Mp, -1 ); 
#else
//      cout << "Multiplying by DDDoe, on device.\n";
        DslashOperatorDDEO(Mp, MMp, 1 );
//      cout << "Multiplying by DDDeo, on device.\n";
        DslashOperatorDDEO(MMp, Mp, -1 );
#endif
   }

   end = clock();

//  gettimeofday(&tim, NULL);  
//  double t2=tim.tv_sec+(tim.tv_usec/1000000.0); 

    
//  hipEventRecord(stop, 0);
//  hipEventSynchronize(stop);
//  hipEventElapsedTime(&time, start, stop);

//  cout << "TEST ran in "<< time / NITERMAX << "ms. \n";
//  cout << "Test run (in "<< (t2-t1)/ NITERMAX<< " sec.)\n";
    cout << "Test run (in "<< (double)(end-start)/( NITERMAX *CLOCKS_PER_SEC) << " sec.)\n";





// this function takes an array on the gpu 
// and puts it into a fermion on the host
   smartUnpackFermionFromDevice(tempFermion,MMp);

#ifndef DOUBLE_PREC
    const char* endFermionFilename = "EndFermionGPU_SinglePrec.fer"; 
#else
    const char* endFermionFilename = "EndFermionGPU_DoublePrec.fer";
#endif
   
    cout << "Saving fermion in " << endFermionFilename << endl; 
    tempFermion->saveToFile(endFermionFilename);
 
    hipFree(MMp);
    hipFree(Mp);
#ifdef DOUBLE_PREC
    hipFree(startFermGPU);
#endif
    cuda_end();
 
    delete tempFermion;
    return 0;
 
}
