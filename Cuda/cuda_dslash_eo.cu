#include "hip/hip_runtime.h"
// SINGLE PRECISION KERNEL for even/odd fermions

__global__ void switchParityKernel(float2 *out,
                                     float2 *in)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x ; 

    float2 temp;
    //1st float
    for(int color = 0 ; color < 3 ; color++){
        temp = in[idx+(color*2+1)*size_dev_h];
    	out[idx+(color*2+1)*size_dev_h] = in[idx+color*2*size_dev_h];
        out[idx+color*2*size_dev_h] = temp ;
    }
    //2nd float
    for(int color = 3 ; color < 6 ; color++){
        temp = in[idx+(color*2+1)*size_dev_h];
    	out[idx+(color*2+1)*size_dev_h] = in[idx+color*2*size_dev_h];
        out[idx+color*2*size_dev_h] = temp ;
    }



}

__global__ void copyKernel(float2 *out,
                                     float2 *in)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x ; 

    
    //1st float
    for(int color = 0 ; color < 3 ; color++){
        out[idx+color*2*size_dev_h] = in[idx+color*2*size_dev_h];
        out[idx+(color*2+1)*size_dev_h] = in[idx+(color*2+1)*size_dev_h];
    }
    //2nd float
    for(int color = 3 ; color < 6 ; color++){
        out[idx+color*2*size_dev_h] = in[idx+color*2*size_dev_h];
        out[idx+(color*2+1)*size_dev_h] = in[idx+(color*2+1)*size_dev_h];
    }

}



__global__ void DslashKernelEO(float2  *out,
                                 float2  *in,
                                 int *tables, 
                                 int *phases, 
                                 size_t gauge_offset)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x + size_dev_h;  // idx>sizeh, ODD

  float stag_phase = 1.0;
  //if(idx < 192) return;

  //Store result in sharedMem
  __shared__ float ferm_out[3][2][NUM_THREADS];

  //New tables indexing (index fastest)
  __shared__ int site_table[NUM_THREADS];

  //Load link matrix U_mu(ix) in registers
  float link0x, link0y, link0z, link0w, 
         link1x, link1y, link1z, link1w, 
         link2x, link2y, link2z, link2w;   
  float4 auxlink;

  float2 ferm_in_0, ferm_in_1, ferm_in_2;


  // DIRECTION 0
  site_table[threadIdx.x]  = tables[idx+4*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];

  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(0+3*0));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(1+3*0));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(2+3*0));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;
  

  ferm_out[0][0][threadIdx.x] = link0x*ferm_in_0.x-link0y*ferm_in_0.y+  
                                link0z*ferm_in_1.x-link0w*ferm_in_1.y+ 
                                link1x*ferm_in_2.x-link1y*ferm_in_2.y; 
  ferm_out[0][1][threadIdx.x] = link0x*ferm_in_0.y+link0y*ferm_in_0.x+ 
                                link0z*ferm_in_1.y+link0w*ferm_in_1.x+ 
                                link1x*ferm_in_2.y+link1y*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] = link1z*ferm_in_0.x-link1w*ferm_in_0.y+  
                                link2x*ferm_in_1.x-link2y*ferm_in_1.y+ 
                                link2z*ferm_in_2.x-link2w*ferm_in_2.y; 
  ferm_out[1][1][threadIdx.x] = link1z*ferm_in_0.y+link1w*ferm_in_0.x+ 
                                link2x*ferm_in_1.y+link2y*ferm_in_1.x+ 
                                link2z*ferm_in_2.y+link2w*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] = C1RED*ferm_in_0.x-C1IMD*ferm_in_0.y+  
                                C2RED*ferm_in_1.x-C2IMD*ferm_in_1.y+ 
                                C3RED*ferm_in_2.x-C3IMD*ferm_in_2.y; 
  ferm_out[2][1][threadIdx.x] = C1RED*ferm_in_0.y+C1IMD*ferm_in_0.x+ 
                                C2RED*ferm_in_1.y+C2IMD*ferm_in_1.x+ 
                                C3RED*ferm_in_2.y+C3IMD*ferm_in_2.x; 

  //DIRECTION 1
  site_table[threadIdx.x] = tables[idx+5*size_dev];
  stag_phase              = (float) phases[idx+size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(0+3*1));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(1+3*1));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(2+3*1));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;
  

ferm_out[0][0][threadIdx.x] += link0x*ferm_in_0.x-link0y*ferm_in_0.y+  
                                 link0z*ferm_in_1.x-link0w*ferm_in_1.y+ 
                                 link1x*ferm_in_2.x-link1y*ferm_in_2.y; 
  ferm_out[0][1][threadIdx.x] += link0x*ferm_in_0.y+link0y*ferm_in_0.x+ 
                                 link0z*ferm_in_1.y+link0w*ferm_in_1.x+ 
                                 link1x*ferm_in_2.y+link1y*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] += link1z*ferm_in_0.x-link1w*ferm_in_0.y+  
                                 link2x*ferm_in_1.x-link2y*ferm_in_1.y+ 
                                 link2z*ferm_in_2.x-link2w*ferm_in_2.y; 
  ferm_out[1][1][threadIdx.x] += link1z*ferm_in_0.y+link1w*ferm_in_0.x+ 
                                 link2x*ferm_in_1.y+link2y*ferm_in_1.x+ 
                                 link2z*ferm_in_2.y+link2w*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.x-C1IMD*ferm_in_0.y+  
					     C2RED*ferm_in_1.x-C2IMD*ferm_in_1.y+ 
					     C3RED*ferm_in_2.x-C3IMD*ferm_in_2.y); 
  ferm_out[2][1][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.y+C1IMD*ferm_in_0.x+ 
					     C2RED*ferm_in_1.y+C2IMD*ferm_in_1.x+ 
					     C3RED*ferm_in_2.y+C3IMD*ferm_in_2.x); 


  //DIRECTION 2
  site_table[threadIdx.x] = tables[idx+6*size_dev];
  stag_phase              = (float) phases[idx+2*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(0+3*2));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(1+3*2));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(2+3*2));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;

  ferm_out[0][0][threadIdx.x] += link0x*ferm_in_0.x-link0y*ferm_in_0.y+  
                                 link0z*ferm_in_1.x-link0w*ferm_in_1.y+ 
                                 link1x*ferm_in_2.x-link1y*ferm_in_2.y; 
  ferm_out[0][1][threadIdx.x] += link0x*ferm_in_0.y+link0y*ferm_in_0.x+ 
                                 link0z*ferm_in_1.y+link0w*ferm_in_1.x+ 
                                 link1x*ferm_in_2.y+link1y*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] += link1z*ferm_in_0.x-link1w*ferm_in_0.y+  
                                 link2x*ferm_in_1.x-link2y*ferm_in_1.y+ 
                                 link2z*ferm_in_2.x-link2w*ferm_in_2.y; 
  ferm_out[1][1][threadIdx.x] += link1z*ferm_in_0.y+link1w*ferm_in_0.x+ 
                                 link2x*ferm_in_1.y+link2y*ferm_in_1.x+ 
                                 link2z*ferm_in_2.y+link2w*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.x-C1IMD*ferm_in_0.y+  
					     C2RED*ferm_in_1.x-C2IMD*ferm_in_1.y+ 
					     C3RED*ferm_in_2.x-C3IMD*ferm_in_2.y); 
  ferm_out[2][1][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.y+C1IMD*ferm_in_0.x+ 
					     C2RED*ferm_in_1.y+C2IMD*ferm_in_1.x+ 
					     C3RED*ferm_in_2.y+C3IMD*ferm_in_2.x); 



  //DIRECTION 3
  site_table[threadIdx.x]  = tables[idx+7*size_dev];
  stag_phase               = (float) phases[idx+3*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(0+3*3));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(1+3*3));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(2+3*3));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;

  ferm_out[0][0][threadIdx.x] += link0x*ferm_in_0.x-link0y*ferm_in_0.y+  
                                 link0z*ferm_in_1.x-link0w*ferm_in_1.y+ 
                                 link1x*ferm_in_2.x-link1y*ferm_in_2.y; 
  ferm_out[0][1][threadIdx.x] += link0x*ferm_in_0.y+link0y*ferm_in_0.x+ 
                                 link0z*ferm_in_1.y+link0w*ferm_in_1.x+ 
                                 link1x*ferm_in_2.y+link1y*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] += link1z*ferm_in_0.x-link1w*ferm_in_0.y+  
                                 link2x*ferm_in_1.x-link2y*ferm_in_1.y+ 
                                 link2z*ferm_in_2.x-link2w*ferm_in_2.y; 
  ferm_out[1][1][threadIdx.x] += link1z*ferm_in_0.y+link1w*ferm_in_0.x+ 
                                 link2x*ferm_in_1.y+link2y*ferm_in_1.x+ 
                                 link2z*ferm_in_2.y+link2w*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.x-C1IMD*ferm_in_0.y+  
					     C2RED*ferm_in_1.x-C2IMD*ferm_in_1.y+ 
					     C3RED*ferm_in_2.x-C3IMD*ferm_in_2.y); 
  ferm_out[2][1][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.y+C1IMD*ferm_in_0.x+ 
					     C2RED*ferm_in_1.y+C2IMD*ferm_in_1.x+ 
					     C3RED*ferm_in_2.y+C3IMD*ferm_in_2.x); 
  

  
  //---------------------------------------------------end of first block

  //DIRECTION 0
  site_table[threadIdx.x] = tables[idx];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(0+3*0));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(1+3*0));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(2+3*0));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;

  ferm_out[0][0][threadIdx.x] -= link0x*ferm_in_0.x+link0y*ferm_in_0.y +
              			 link1z*ferm_in_1.x+link1w*ferm_in_1.y +
				 C1RED*ferm_in_2.x   +C1IMD*ferm_in_2.y; 
  
  ferm_out[0][1][threadIdx.x] -= link0x*ferm_in_0.y-link0y*ferm_in_0.x +
                                 link1z*ferm_in_1.y-link1w*ferm_in_1.x +
                                 C1RED*ferm_in_2.y   -C1IMD*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] -= link0z*ferm_in_0.x+link0w*ferm_in_0.y +
                                 link2x*ferm_in_1.x+link2y*ferm_in_1.y +
                                 C2RED*ferm_in_2.x   +C2IMD*ferm_in_2.y; 

  ferm_out[1][1][threadIdx.x] -= link0z*ferm_in_0.y-link0w*ferm_in_0.x +
                                 link2x*ferm_in_1.y-link2y*ferm_in_1.x +
                                 C2RED*ferm_in_2.y   -C2IMD*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] -= link1x*ferm_in_0.x+link1y*ferm_in_0.y +
                                 link2z*ferm_in_1.x+link2w*ferm_in_1.y +
                                 C3RED*ferm_in_2.x   +C3IMD*ferm_in_2.y; 

  ferm_out[2][1][threadIdx.x] -= link1x*ferm_in_0.y-link1y*ferm_in_0.x +
                                 link2z*ferm_in_1.y-link2w*ferm_in_1.x +
                                 C3RED*ferm_in_2.y   -C3IMD*ferm_in_2.x; 



  
  //DIRECTION 1
  site_table[threadIdx.x] = tables[idx+size_dev];
  stag_phase              = (float) phases[site_table[threadIdx.x]+size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(0+3*1));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(1+3*1));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(2+3*1));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] -= link0x*ferm_in_0.x+link0y*ferm_in_0.y +
                                 link1z*ferm_in_1.x+link1w*ferm_in_1.y +
                                 stag_phase*(C1RED*ferm_in_2.x+C1IMD*ferm_in_2.y); 

  ferm_out[0][1][threadIdx.x] -= link0x*ferm_in_0.y-link0y*ferm_in_0.x +
                                 link1z*ferm_in_1.y-link1w*ferm_in_1.x +
                                 stag_phase*(C1RED*ferm_in_2.y-C1IMD*ferm_in_2.x); 

  ferm_out[1][0][threadIdx.x] -= link0z*ferm_in_0.x+link0w*ferm_in_0.y +
                                 link2x*ferm_in_1.x+link2y*ferm_in_1.y +
                                 stag_phase*(C2RED*ferm_in_2.x+C2IMD*ferm_in_2.y); 

  ferm_out[1][1][threadIdx.x] -= link0z*ferm_in_0.y-link0w*ferm_in_0.x +
                                 link2x*ferm_in_1.y-link2y*ferm_in_1.x +
                                 stag_phase*(C2RED*ferm_in_2.y-C2IMD*ferm_in_2.x); 

  ferm_out[2][0][threadIdx.x] -= link1x*ferm_in_0.x+link1y*ferm_in_0.y +
                                 link2z*ferm_in_1.x+link2w*ferm_in_1.y +
                                 stag_phase*(C3RED*ferm_in_2.x+C3IMD*ferm_in_2.y); 

  ferm_out[2][1][threadIdx.x] -= link1x*ferm_in_0.y-link1y*ferm_in_0.x +
                                 link2z*ferm_in_1.y-link2w*ferm_in_1.x +
                                 stag_phase*(C3RED*ferm_in_2.y- C3IMD*ferm_in_2.x); 



  //DIRECTION 2
  site_table[threadIdx.x] = tables[idx+2*size_dev];
  stag_phase              = (float) phases[site_table[threadIdx.x]+2*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];

  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(0+3*2));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(1+3*2));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(2+3*2));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] -= link0x*ferm_in_0.x+link0y*ferm_in_0.y +
                                 link1z*ferm_in_1.x+link1w*ferm_in_1.y +
                                 stag_phase*(C1RED*ferm_in_2.x+ C1IMD*ferm_in_2.y); 

  ferm_out[0][1][threadIdx.x] -= link0x*ferm_in_0.y-link0y*ferm_in_0.x +
                                 link1z*ferm_in_1.y-link1w*ferm_in_1.x +
                                 stag_phase*(C1RED*ferm_in_2.y- C1IMD*ferm_in_2.x); 

  ferm_out[1][0][threadIdx.x] -= link0z*ferm_in_0.x+link0w*ferm_in_0.y +
                                 link2x*ferm_in_1.x+link2y*ferm_in_1.y +
                                 stag_phase*(C2RED*ferm_in_2.x+ C2IMD*ferm_in_2.y); 

  ferm_out[1][1][threadIdx.x] -= link0z*ferm_in_0.y-link0w*ferm_in_0.x +
                                 link2x*ferm_in_1.y-link2y*ferm_in_1.x +
                                 stag_phase*(C2RED*ferm_in_2.y- C2IMD*ferm_in_2.x); 

  ferm_out[2][0][threadIdx.x] -= link1x*ferm_in_0.x+link1y*ferm_in_0.y +
                                 link2z*ferm_in_1.x+link2w*ferm_in_1.y +
                                 stag_phase*(C3RED*ferm_in_2.x+ C3IMD*ferm_in_2.y); 

  ferm_out[2][1][threadIdx.x] -= link1x*ferm_in_0.y-link1y*ferm_in_0.x +
                                 link2z*ferm_in_1.y-link2w*ferm_in_1.x +
                                 stag_phase*(C3RED*ferm_in_2.y- C3IMD*ferm_in_2.x); 



  //DIRECTION 3
  site_table[threadIdx.x] = tables[idx+3*size_dev];
  stag_phase              = (float) phases[site_table[threadIdx.x]+3*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(0+3*3));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(1+3*3));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(2+3*3));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] -= link0x*ferm_in_0.x+link0y*ferm_in_0.y +
                                 link1z*ferm_in_1.x+link1w*ferm_in_1.y +
                                 stag_phase*(C1RED*ferm_in_2.x+  C1IMD*ferm_in_2.y); 

  ferm_out[0][1][threadIdx.x] -= link0x*ferm_in_0.y-link0y*ferm_in_0.x +
                                 link1z*ferm_in_1.y-link1w*ferm_in_1.x +
                                 stag_phase*(C1RED*ferm_in_2.y- C1IMD*ferm_in_2.x); 

  ferm_out[1][0][threadIdx.x] -= link0z*ferm_in_0.x+link0w*ferm_in_0.y +
                                 link2x*ferm_in_1.x+link2y*ferm_in_1.y +
                                 stag_phase*(C2RED*ferm_in_2.x+ C2IMD*ferm_in_2.y); 

  ferm_out[1][1][threadIdx.x] -= link0z*ferm_in_0.y-link0w*ferm_in_0.x +
                                 link2x*ferm_in_1.y-link2y*ferm_in_1.x +
                                 stag_phase*(C2RED*ferm_in_2.y- C2IMD*ferm_in_2.x); 

  ferm_out[2][0][threadIdx.x] -= link1x*ferm_in_0.x+link1y*ferm_in_0.y +
                                 link2z*ferm_in_1.x+link2w*ferm_in_1.y +
                                 stag_phase*(C3RED*ferm_in_2.x+ C3IMD*ferm_in_2.y); 

  ferm_out[2][1][threadIdx.x] -= link1x*ferm_in_0.y-link1y*ferm_in_0.x +
                                 link2z*ferm_in_1.y-link2w*ferm_in_1.x +
                                 stag_phase*(C3RED*ferm_in_2.y- C3IMD*ferm_in_2.x); 
  
  //-------------------------------------------------end of second block

  // even
  ferm_in_0 = in[              idx - size_dev_h];
  ferm_in_1 = in[   size_dev + idx - size_dev_h];
  ferm_in_2 = in[ 2*size_dev + idx - size_dev_h];

//  out[idx              - size_dev_h ].x = 0 ; // mass_d_dev*ferm_in_0.x;
//  out[idx              - size_dev_h ].y = 0 ; // mass_d_dev*ferm_in_0.y;
//  out[idx +   size_dev - size_dev_h ].x = 0 ; // mass_d_dev*ferm_in_1.x;
//  out[idx +   size_dev - size_dev_h ].y = 0 ; // mass_d_dev*ferm_in_1.y;
//  out[idx + 2*size_dev - size_dev_h ].x = 0 ; // mass_d_dev*ferm_in_2.x;
//  out[idx + 2*size_dev - size_dev_h ].y = 0 ; // mass_d_dev*ferm_in_2.y;

  //odd
  out[idx               ].x =ferm_out[0][0][threadIdx.x]*(float)0.5;
  out[idx               ].y =ferm_out[0][1][threadIdx.x]*(float)0.5;
  out[idx +   size_dev  ].x =ferm_out[1][0][threadIdx.x]*(float)0.5;
  out[idx +   size_dev  ].y =ferm_out[1][1][threadIdx.x]*(float)0.5;
  out[idx + 2*size_dev  ].x =ferm_out[2][0][threadIdx.x]*(float)0.5;
  out[idx + 2*size_dev  ].y =ferm_out[2][1][threadIdx.x]*(float)0.5;

  //-------------------------------------------------end of Dslash
  }







__global__ void DslashDaggerKernelEO(float2 *out,
                                       float2 *in,
                                       int *tables, 
                                       int *phases,
                                       size_t gauge_offset) 
  { 
  int idx = blockIdx.x*blockDim.x + threadIdx.x;     // idx< sizeh, EVEN!!
  float stag_phase = 1.0;

  //Store result in sharedMem
  __shared__ float ferm_out[3][2][NUM_THREADS];
 
  //New tables indexing (index fastest)
  __shared__ int site_table[NUM_THREADS];

  //Load link matrix U_mu(ix) in registers
  float link0x, link0y, link0z, link0w, 
         link1x, link1y, link1z, link1w, 
         link2x, link2y, link2z, link2w;   
  float4 auxlink;

  float2 ferm_in_0, ferm_in_1, ferm_in_2;
  
  // DIRECTION 0
  site_table[threadIdx.x] = tables[idx+4*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];



 
  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(0+3*0));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(1+3*0));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(2+3*0));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] = link0x*ferm_in_0.x-link0y*ferm_in_0.y+  
                                link0z*ferm_in_1.x-link0w*ferm_in_1.y+ 
                                link1x*ferm_in_2.x-link1y*ferm_in_2.y; 
  ferm_out[0][1][threadIdx.x] = link0x*ferm_in_0.y+link0y*ferm_in_0.x+ 
                                link0z*ferm_in_1.y+link0w*ferm_in_1.x+ 
                                link1x*ferm_in_2.y+link1y*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] = link1z*ferm_in_0.x-link1w*ferm_in_0.y+  
                                link2x*ferm_in_1.x-link2y*ferm_in_1.y+ 
                                link2z*ferm_in_2.x-link2w*ferm_in_2.y; 
  ferm_out[1][1][threadIdx.x] = link1z*ferm_in_0.y+link1w*ferm_in_0.x+ 
                                link2x*ferm_in_1.y+link2y*ferm_in_1.x+ 
                                link2z*ferm_in_2.y+link2w*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] = C1RED*ferm_in_0.x-C1IMD*ferm_in_0.y+  
                                C2RED*ferm_in_1.x-C2IMD*ferm_in_1.y+ 
                                C3RED*ferm_in_2.x-C3IMD*ferm_in_2.y; 
  ferm_out[2][1][threadIdx.x] = C1RED*ferm_in_0.y+C1IMD*ferm_in_0.x+ 
                                C2RED*ferm_in_1.y+C2IMD*ferm_in_1.x+ 
                                C3RED*ferm_in_2.y+C3IMD*ferm_in_2.x; 



  //DIRECTION 1
  site_table[threadIdx.x] = tables[idx+5*size_dev];
  stag_phase              = (float) phases[idx+size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(0+3*1));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(1+3*1));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(2+3*1));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] += link0x*ferm_in_0.x-link0y*ferm_in_0.y+  
                                 link0z*ferm_in_1.x-link0w*ferm_in_1.y+ 
                                 link1x*ferm_in_2.x-link1y*ferm_in_2.y; 
  ferm_out[0][1][threadIdx.x] += link0x*ferm_in_0.y+link0y*ferm_in_0.x+ 
                                 link0z*ferm_in_1.y+link0w*ferm_in_1.x+ 
                                 link1x*ferm_in_2.y+link1y*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] += link1z*ferm_in_0.x-link1w*ferm_in_0.y+  
                                 link2x*ferm_in_1.x-link2y*ferm_in_1.y+ 
                                 link2z*ferm_in_2.x-link2w*ferm_in_2.y; 
  ferm_out[1][1][threadIdx.x] += link1z*ferm_in_0.y+link1w*ferm_in_0.x+ 
                                 link2x*ferm_in_1.y+link2y*ferm_in_1.x+ 
                                 link2z*ferm_in_2.y+link2w*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.x-C1IMD*ferm_in_0.y+  
					     C2RED*ferm_in_1.x-C2IMD*ferm_in_1.y+ 
					     C3RED*ferm_in_2.x-C3IMD*ferm_in_2.y); 
  ferm_out[2][1][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.y+C1IMD*ferm_in_0.x+ 
					     C2RED*ferm_in_1.y+C2IMD*ferm_in_1.x+ 
					     C3RED*ferm_in_2.y+C3IMD*ferm_in_2.x); 
   


  //DIRECTION 2
  site_table[threadIdx.x] = tables[idx+6*size_dev];
  stag_phase              = (float) phases[idx+2*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];

  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(0+3*2));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(1+3*2));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(2+3*2));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] += link0x*ferm_in_0.x-link0y*ferm_in_0.y+  
                                 link0z*ferm_in_1.x-link0w*ferm_in_1.y+ 
                                 link1x*ferm_in_2.x-link1y*ferm_in_2.y; 
  ferm_out[0][1][threadIdx.x] += link0x*ferm_in_0.y+link0y*ferm_in_0.x+ 
                                 link0z*ferm_in_1.y+link0w*ferm_in_1.x+ 
                                 link1x*ferm_in_2.y+link1y*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] += link1z*ferm_in_0.x-link1w*ferm_in_0.y+  
                                 link2x*ferm_in_1.x-link2y*ferm_in_1.y+ 
                                 link2z*ferm_in_2.x-link2w*ferm_in_2.y; 
  ferm_out[1][1][threadIdx.x] += link1z*ferm_in_0.y+link1w*ferm_in_0.x+ 
                                 link2x*ferm_in_1.y+link2y*ferm_in_1.x+ 
                                 link2z*ferm_in_2.y+link2w*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.x-C1IMD*ferm_in_0.y+  
					     C2RED*ferm_in_1.x-C2IMD*ferm_in_1.y+ 
					     C3RED*ferm_in_2.x-C3IMD*ferm_in_2.y); 
  ferm_out[2][1][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.y+C1IMD*ferm_in_0.x+ 
					     C2RED*ferm_in_1.y+C2IMD*ferm_in_1.x+ 
					     C3RED*ferm_in_2.y+C3IMD*ferm_in_2.x); 

  
  //DIRECTION 3
  site_table[threadIdx.x] = tables[idx+7*size_dev];
   stag_phase              = (float) phases[idx+3*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(0+3*3));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(1+3*3));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, idx + gauge_offset + size_dev*(2+3*3));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;

  ferm_out[0][0][threadIdx.x] += link0x*ferm_in_0.x-link0y*ferm_in_0.y+  
                                 link0z*ferm_in_1.x-link0w*ferm_in_1.y+ 
                                 link1x*ferm_in_2.x-link1y*ferm_in_2.y; 
  ferm_out[0][1][threadIdx.x] += link0x*ferm_in_0.y+link0y*ferm_in_0.x+ 
                                 link0z*ferm_in_1.y+link0w*ferm_in_1.x+ 
                                 link1x*ferm_in_2.y+link1y*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] += link1z*ferm_in_0.x-link1w*ferm_in_0.y+  
                                 link2x*ferm_in_1.x-link2y*ferm_in_1.y+ 
                                 link2z*ferm_in_2.x-link2w*ferm_in_2.y; 
  ferm_out[1][1][threadIdx.x] += link1z*ferm_in_0.y+link1w*ferm_in_0.x+ 
                                 link2x*ferm_in_1.y+link2y*ferm_in_1.x+ 
                                 link2z*ferm_in_2.y+link2w*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.x-C1IMD*ferm_in_0.y+  
					     C2RED*ferm_in_1.x-C2IMD*ferm_in_1.y+ 
					     C3RED*ferm_in_2.x-C3IMD*ferm_in_2.y); 
  ferm_out[2][1][threadIdx.x] += stag_phase*(C1RED*ferm_in_0.y+C1IMD*ferm_in_0.x+ 
					     C2RED*ferm_in_1.y+C2IMD*ferm_in_1.x+ 
					     C3RED*ferm_in_2.y+C3IMD*ferm_in_2.x); 


  //---------------------------------------------------end of first block
 
  //DIRECTION 0
  site_table[threadIdx.x] = tables[idx];
 
  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];


  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(0+3*0));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(1+3*0));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(2+3*0));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] -= link0x*ferm_in_0.x+link0y*ferm_in_0.y +
              			 link1z*ferm_in_1.x+link1w*ferm_in_1.y +
				 C1RED*ferm_in_2.x   +C1IMD*ferm_in_2.y; 
  
  ferm_out[0][1][threadIdx.x] -= link0x*ferm_in_0.y-link0y*ferm_in_0.x +
                                 link1z*ferm_in_1.y-link1w*ferm_in_1.x +
                                 C1RED*ferm_in_2.y   -C1IMD*ferm_in_2.x; 

  ferm_out[1][0][threadIdx.x] -= link0z*ferm_in_0.x+link0w*ferm_in_0.y +
                                 link2x*ferm_in_1.x+link2y*ferm_in_1.y +
                                 C2RED*ferm_in_2.x   +C2IMD*ferm_in_2.y; 

  ferm_out[1][1][threadIdx.x] -= link0z*ferm_in_0.y-link0w*ferm_in_0.x +
                                 link2x*ferm_in_1.y-link2y*ferm_in_1.x +
                                 C2RED*ferm_in_2.y   -C2IMD*ferm_in_2.x; 

  ferm_out[2][0][threadIdx.x] -= link1x*ferm_in_0.x+link1y*ferm_in_0.y +
                                 link2z*ferm_in_1.x+link2w*ferm_in_1.y +
                                 C3RED*ferm_in_2.x   +C3IMD*ferm_in_2.y; 

  ferm_out[2][1][threadIdx.x] -= link1x*ferm_in_0.y-link1y*ferm_in_0.x +
                                 link2z*ferm_in_1.y-link2w*ferm_in_1.x +
                                 C3RED*ferm_in_2.y   -C3IMD*ferm_in_2.x; 
  

  //DIRECTION 1
  site_table[threadIdx.x] = tables[idx+size_dev];
  stag_phase              = (float) phases[site_table[threadIdx.x]+size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];

  // 1st float 
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(0+3*1));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(1+3*1));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(2+3*1));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] -= link0x*ferm_in_0.x+link0y*ferm_in_0.y +
                                 link1z*ferm_in_1.x+link1w*ferm_in_1.y +
                                 stag_phase*(C1RED*ferm_in_2.x+C1IMD*ferm_in_2.y); 

  ferm_out[0][1][threadIdx.x] -= link0x*ferm_in_0.y-link0y*ferm_in_0.x +
                                 link1z*ferm_in_1.y-link1w*ferm_in_1.x +
                                 stag_phase*(C1RED*ferm_in_2.y-C1IMD*ferm_in_2.x); 

  ferm_out[1][0][threadIdx.x] -= link0z*ferm_in_0.x+link0w*ferm_in_0.y +
                                 link2x*ferm_in_1.x+link2y*ferm_in_1.y +
                                 stag_phase*(C2RED*ferm_in_2.x+C2IMD*ferm_in_2.y); 

  ferm_out[1][1][threadIdx.x] -= link0z*ferm_in_0.y-link0w*ferm_in_0.x +
                                 link2x*ferm_in_1.y-link2y*ferm_in_1.x +
                                 stag_phase*(C2RED*ferm_in_2.y-C2IMD*ferm_in_2.x); 

  ferm_out[2][0][threadIdx.x] -= link1x*ferm_in_0.x+link1y*ferm_in_0.y +
                                 link2z*ferm_in_1.x+link2w*ferm_in_1.y +
                                 stag_phase*(C3RED*ferm_in_2.x+C3IMD*ferm_in_2.y); 

  ferm_out[2][1][threadIdx.x] -= link1x*ferm_in_0.y-link1y*ferm_in_0.x +
                                 link2z*ferm_in_1.y-link2w*ferm_in_1.x +
                                 stag_phase*(C3RED*ferm_in_2.y- C3IMD*ferm_in_2.x); 

  //DIRECTION 2
  site_table[threadIdx.x] = tables[idx+2*size_dev];
  stag_phase              = (float) phases[site_table[threadIdx.x]+2*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];

  // 1st float
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(0+3*2));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(1+3*2));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(2+3*2));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;


  ferm_out[0][0][threadIdx.x] -= link0x*ferm_in_0.x+link0y*ferm_in_0.y +
                                 link1z*ferm_in_1.x+link1w*ferm_in_1.y +
                                 stag_phase*(C1RED*ferm_in_2.x+ C1IMD*ferm_in_2.y); 

  ferm_out[0][1][threadIdx.x] -= link0x*ferm_in_0.y-link0y*ferm_in_0.x +
                                 link1z*ferm_in_1.y-link1w*ferm_in_1.x +
                                 stag_phase*(C1RED*ferm_in_2.y- C1IMD*ferm_in_2.x); 

  ferm_out[1][0][threadIdx.x] -= link0z*ferm_in_0.x+link0w*ferm_in_0.y +
                                 link2x*ferm_in_1.x+link2y*ferm_in_1.y +
                                 stag_phase*(C2RED*ferm_in_2.x+ C2IMD*ferm_in_2.y); 

  ferm_out[1][1][threadIdx.x] -= link0z*ferm_in_0.y-link0w*ferm_in_0.x +
                                 link2x*ferm_in_1.y-link2y*ferm_in_1.x +
                                 stag_phase*(C2RED*ferm_in_2.y- C2IMD*ferm_in_2.x); 

  ferm_out[2][0][threadIdx.x] -= link1x*ferm_in_0.x+link1y*ferm_in_0.y +
                                 link2z*ferm_in_1.x+link2w*ferm_in_1.y +
                                 stag_phase*(C3RED*ferm_in_2.x+ C3IMD*ferm_in_2.y); 

  ferm_out[2][1][threadIdx.x] -= link1x*ferm_in_0.y-link1y*ferm_in_0.x +
                                 link2z*ferm_in_1.y-link2w*ferm_in_1.x +
                                 stag_phase*(C3RED*ferm_in_2.y- C3IMD*ferm_in_2.x); 



  //DIRECTION 3

 site_table[threadIdx.x] = tables[idx+3*size_dev];
  stag_phase              = (float) phases[site_table[threadIdx.x]+3*size_dev];

  ferm_in_0 = in[              site_table[threadIdx.x]];
  ferm_in_1 = in[   size_dev + site_table[threadIdx.x]];
  ferm_in_2 = in[ 2*size_dev + site_table[threadIdx.x]];

  // 1st float
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(0+3*3));
  link0x=(float) auxlink.x;
  link0y=(float) auxlink.y;
  link0z=(float) auxlink.z;
  link0w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(1+3*3));
  link1x=(float) auxlink.x;
  link1y=(float) auxlink.y;
  link1z=(float) auxlink.z;
  link1w=(float) auxlink.w;
  auxlink = tex1Dfetch(gauge_texRef, site_table[threadIdx.x] + gauge_offset + size_dev*(2+3*3));
  link2x=(float) auxlink.x;
  link2y=(float) auxlink.y;
  link2z=(float) auxlink.z;
  link2w=(float) auxlink.w;

  ferm_out[0][0][threadIdx.x] -= link0x*ferm_in_0.x+link0y*ferm_in_0.y +
                                 link1z*ferm_in_1.x+link1w*ferm_in_1.y +
                                 stag_phase*(C1RED*ferm_in_2.x+  C1IMD*ferm_in_2.y); 

  ferm_out[0][1][threadIdx.x] -= link0x*ferm_in_0.y-link0y*ferm_in_0.x +
                                 link1z*ferm_in_1.y-link1w*ferm_in_1.x +
                                 stag_phase*(C1RED*ferm_in_2.y- C1IMD*ferm_in_2.x); 

  ferm_out[1][0][threadIdx.x] -= link0z*ferm_in_0.x+link0w*ferm_in_0.y +
                                 link2x*ferm_in_1.x+link2y*ferm_in_1.y +
                                 stag_phase*(C2RED*ferm_in_2.x+ C2IMD*ferm_in_2.y); 

  ferm_out[1][1][threadIdx.x] -= link0z*ferm_in_0.y-link0w*ferm_in_0.x +
                                 link2x*ferm_in_1.y-link2y*ferm_in_1.x +
                                 stag_phase*(C2RED*ferm_in_2.y- C2IMD*ferm_in_2.x); 

  ferm_out[2][0][threadIdx.x] -= link1x*ferm_in_0.x+link1y*ferm_in_0.y +
                                 link2z*ferm_in_1.x+link2w*ferm_in_1.y +
                                 stag_phase*(C3RED*ferm_in_2.x+ C3IMD*ferm_in_2.y); 

  ferm_out[2][1][threadIdx.x] -= link1x*ferm_in_0.y-link1y*ferm_in_0.x +
                                 link2z*ferm_in_1.y-link2w*ferm_in_1.x +
                                 stag_phase*(C3RED*ferm_in_2.y- C3IMD*ferm_in_2.x); 

  //-------------------------------------------------end of second block

  // even   
  ferm_in_0 = in[              idx];
  ferm_in_1 = in[   size_dev + idx];
  ferm_in_2 = in[ 2*size_dev + idx];
/*
  out[idx               ].x = mass_d_dev*ferm_in_0.x - ferm_out[0][0][threadIdx.x]*(float)0.5;
  out[idx               ].y = mass_d_dev*ferm_in_0.y - ferm_out[0][1][threadIdx.x]*(float)0.5;
  out[idx +   size_dev  ].x = mass_d_dev*ferm_in_1.x - ferm_out[1][0][threadIdx.x]*(float)0.5;
  out[idx +   size_dev  ].y = mass_d_dev*ferm_in_1.y - ferm_out[1][1][threadIdx.x]*(float)0.5;
  out[idx + 2*size_dev  ].x = mass_d_dev*ferm_in_2.x - ferm_out[2][0][threadIdx.x]*(float)0.5;
  out[idx + 2*size_dev  ].y = mass_d_dev*ferm_in_2.y - ferm_out[2][1][threadIdx.x]*(float)0.5;
*/
  out[idx               ].x =ferm_out[0][0][threadIdx.x]*(float)0.5;
  out[idx               ].y =ferm_out[0][1][threadIdx.x]*(float)0.5;
  out[idx +   size_dev  ].x =ferm_out[1][0][threadIdx.x]*(float)0.5;
  out[idx +   size_dev  ].y =ferm_out[1][1][threadIdx.x]*(float)0.5;
  out[idx + 2*size_dev  ].x =ferm_out[2][0][threadIdx.x]*(float)0.5;
  out[idx + 2*size_dev  ].y =ferm_out[2][1][threadIdx.x]*(float)0.5;


  // odd
  out[idx              + size_dev_h ].x = (float)0.0;
  out[idx              + size_dev_h ].y = (float)0.0;
  out[idx +   size_dev + size_dev_h ].x = (float)0.0;
  out[idx +   size_dev + size_dev_h ].y = (float)0.0;
  out[idx + 2*size_dev + size_dev_h ].x = (float)0.0;
  out[idx + 2*size_dev + size_dev_h ].y = (float)0.0;

  //-------------------------------------------------end of DslashDagger
  }






/*
================================================================= EXTERNAL C FUNCTION
*/

void switchParity(float2 *out, float2* in)
{

  dim3 BlockDimension(NUM_THREADS);
  dim3 GridDimension(sizeh/BlockDimension.x);  //Half sites

  switchParityKernel<<<GridDimension,BlockDimension>>>(out,in);


}
void copy(float2 *out, float2* in)
{

  dim3 BlockDimension(NUM_THREADS);
  dim3 GridDimension(sizeh/BlockDimension.x);  //Half sites

  copyKernel<<<GridDimension,BlockDimension>>>(out,in);


}

void DslashOperatorEO(float2 *out, 
 		        float2 *in, 
 		        const int isign)
  {
  #ifdef DEBUG_MODE_2
  printf("\033[32mDEBUG: inside DslashOperatorDDEO ...\033[0m\n");
  #endif

  dim3 BlockDimension(NUM_THREADS);
  dim3 GridDimension(sizeh/BlockDimension.x);  //Half sites

  size_t gauge_field_size = sizeof(float4)*size*12;

  size_t offset_g;
  cudaSafe(AT,hipBindTexture(&offset_g, gauge_texRef, gauge_field_device, 2*gauge_field_size), "hipBindTexture");  
  offset_g/=sizeof(float4);

  if(isign == PLUS) 
    {
    DslashKernelEO<<<GridDimension,BlockDimension>>>(out, in, device_table, device_phases, offset_g); 
    cudaCheckError(AT,"DslashDDKernelEO"); 
    }
  
  if(isign == MINUS) 
    {
    DslashDaggerKernelEO<<<GridDimension,BlockDimension>>>(out, in, device_table, device_phases, offset_g); 
    cudaCheckError(AT,"DslashDaggerDDKernelEO"); 
    }

  cudaSafe(AT,hipUnbindTexture(gauge_texRef), "hipUnbindTexture");

  #ifdef DEBUG_MODE_2
  printf("\033[32m\tterminated DslashOperatorDDEO \033[0m\n");
  #endif
  }

