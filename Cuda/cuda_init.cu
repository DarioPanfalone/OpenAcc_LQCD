#include "hip/hip_runtime.h"

extern "C" void cuda_init0(void)
  {
  #ifdef DEBUG_MODE
  printf("DEBUG: inside cuda_init0 ...\n");
  #endif

  size_t gauge_field_size_f = 3*sizeof(float4)*no_links;    //first two lines only of each SU(3) matrix

  // allocate & initialize gauge
  // 2 since 1double~2float

  cudaSafe(AT,hipMalloc((void**)&gauge_field_device, 2*gauge_field_size_f), "hipMalloc");
  cudaSafe(AT,hipMemcpy(gauge_field_device, gauge_field_packed, 2*gauge_field_size_f, hipMemcpyHostToDevice), 
                 "hipMemcpy");


  // allocate & initialize device_table
  cudaSafe(AT,hipMalloc((void**)&device_table, sizeof(int)*size*8), "hipMalloc");
  cudaSafe(AT,hipMemcpy(device_table, shift_table, sizeof(int)*size*8, hipMemcpyHostToDevice), "hipMemcpy"); 

  // allocate & initialize device_phases
  cudaSafe(AT,hipMalloc((void**)&device_phases, sizeof(int)*size*4), "hipMalloc");
  cudaSafe(AT,hipMemcpy(device_phases, eta, sizeof(int)*size*4, hipMemcpyHostToDevice), "hipMemcpy"); 


  // initialize constants
  float mass_l=(float) mass;
  cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(mass_dev), &mass_l, sizeof(float), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
  cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(mass_d_dev), &mass, sizeof(double), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
  int size_l=(int) size;
  cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(size_dev), &size_l, sizeof(int), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
  size_l=(int) sizeh;
  cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(size_dev_h), &size_l, sizeof(int), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");


  #ifdef DEBUG_MODE
  printf("\tterminated cuda_init0\n");
  #endif
  }


extern "C" void cuda_init1(void)
  {
  #ifdef DEBUG_MODE
  printf("DEBUG: inside cuda_init1 ...\n");
  #endif

  size_t vector_size_f   = sizeof(float2)*3*size;           // 2(complex)*3(su3_vector)

  // allocate & initialize mf_device
  // again 2 since 1double~2float
  cudaSafe(AT,hipMalloc((void**)&mf_device, 2*vector_size_f), "hipMalloc"); 
  cudaSafe(AT,hipMemset(mf_device, 0, 2*vector_size_f), "hipMemset");  // initialize even and odd to 0
     
     // 1st float
     cudaSafe(AT,hipMemcpy(mf_device , chi_packed , 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
     cudaSafe(AT,hipMemcpy(mf_device +   size , chi_packed +   size , 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
     cudaSafe(AT,hipMemcpy(mf_device + 2*size , chi_packed + 2*size , 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");

     // 2nd float
     cudaSafe(AT,hipMemcpy(mf_device +          3*size, chi_packed +          3*size,  size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
     cudaSafe(AT,hipMemcpy(mf_device +   size + 3*size, chi_packed +   size + 3*size,  size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
     cudaSafe(AT,hipMemcpy(mf_device + 2*size + 3*size, chi_packed + 2*size + 3*size,  size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");

     

  // allocate & initialize to zero smf_device (even & odd)
  // again 2 since 1double~2float
  cudaSafe(AT,hipMalloc((void**)&smf_device, 2*vector_size_f), "hipMalloc"); 
  cudaSafe(AT,hipMemset(smf_device, 0, 2*vector_size_f), "hipMemset"); 

  #ifdef DEBUG_MODE
  printf("\tterminated cuda_init1\n");
  #endif
  }


extern "C" void cuda_end(void)
  {
  #ifdef DEBUG_MODE
  printf("DEBUG: inside cuda_end ...\n");
  #endif

  cudaSafe(AT,hipMemcpy(gauge_field_packed, gauge_field_device, 2*3*no_links*sizeof(float4), hipMemcpyDeviceToHost), 
                                                                                                            "hipMemcpy");

  cudaSafe(AT,hipFree(gauge_field_device), "hipFree");
  cudaSafe(AT,hipFree(device_table), "hipFree");
  cudaSafe(AT,hipFree(device_phases), "hipFree");

  cudaSafe(AT,hipFree(mf_device), "hipFree");
  cudaSafe(AT,hipFree(smf_device), "hipFree");

  #ifdef DEBUG_MODE
  printf("\tterminated cuda_end\n");
  #endif
  }


extern "C" void cuda_get_conf(void)
 {
 #ifdef DEBUG_MODE
 printf("DEBUG: inside cuda_get_conf ...\n");
 #endif

 cudaSafe(AT,hipMemcpy(gauge_field_packed, gauge_field_device, 2*12*no_links*sizeof(float), hipMemcpyDeviceToHost), 
       "hipMemcpy");

 #ifdef DEBUG_MODE
 printf("\tterminated cuda_get_conf ...\n");
 #endif
 }


